#include "hip/hip_runtime.h"
#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>  // for high_resolution_clock

#include "cpu_solver.h"
#include "gpu_solver.cuh"
#include "utils.h"

void dry_run(int N){

        std::cout << "######################################################" << std::endl;
        std::cout << "A quick test to evaluate overall functionality and performance" << std::endl;
        std::cout << "######################################################" << std::endl;

        int testruns = 10;

        float *A, *B, *C, *D, *E;
        float *min;

        // host malloc

        checkCuda(hipHostMalloc((void**)&A, N * sizeof(float))); // host pinned
        checkCuda(hipHostMalloc((void**)&B, N * sizeof(float))); // host pinned
        checkCuda(hipHostMalloc((void**)&C, N * sizeof(float))); // host pinned
        checkCuda(hipHostMalloc((void**)&D, N * sizeof(float))); // host pinned
        checkCuda(hipHostMalloc((void**)&E, N * sizeof(float))); // host pinned

        checkCuda(hipHostMalloc((void**)&min, N * sizeof(float))); // host pinned

        memset(min, 0, N * sizeof(float));

        std::cout << "generating data..." << std::endl;

        generate_data(N, -100, 100, A);
        generate_data(N, -100, 100, B);
        generate_data(N, -100, 100, C);
        generate_data(N, -100, 100, D);
        generate_data(N, -100, 100, E);

        for (int i = 0; i < N; i++) {
                if (A[i] == 0) { A[i] = 1; } // done to avoid undefined behaviour in solver when A=0
        }

        std::cout << "done!" << std::endl;

        float dur = 0;
        float milliseconds = 0;
        float avg_cpu = 0;
        float avg = 0;

        std::cout << "####################### CPU ##########################" << std::endl;
        std::cout << "######################################################" << std::endl;
        std::cout << "######################################################" << std::endl;

        for (int k = 0; k < testruns; ++k) {
                auto pstart = std::chrono::high_resolution_clock::now();

                QuarticMinimumCPU(N, A, B, C, D, E, min);

                auto finish = std::chrono::high_resolution_clock::now();
                std::chrono::duration<float> elapsed = finish - pstart;
                dur = elapsed.count() * 1000;
                printf("Time (ms): %f\n", dur);
                avg_cpu += dur;
        }

        printf("min[0]: %f \n",min[0]);
        printf("avgTime (ms): %f\n", avg_cpu / testruns);

        memset(min, 0, N * sizeof(float));

        avg = 0;

        std::cout << "####################### GPU (no streams) #############" << std::endl;
        std::cout << "######################################################" << std::endl;
        std::cout << "######################################################" << std::endl;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        for (int k = 0; k < testruns; ++k) {

                hipEventRecord(start);

                QuarticMinimumGPU(N, A, B, C, D, E, min);

                hipEventRecord(stop);

                hipEventSynchronize(stop);
                milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                printf("Time (ms): %f\n", milliseconds);
                avg += milliseconds;
        }

        printf("min[0]: %f \n",min[0]);
        printf("avgTime (ms): %f\n", avg / testruns);

        avg = 0;

        std::cout << "####################### GPU (streams) ################" << std::endl;
        std::cout << "######################################################" << std::endl;
        std::cout << "######################################################" << std::endl;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        for (int k = 0; k < testruns; ++k) {

                hipEventRecord(start);

                QuarticMinimumGPUStreams(N, A, B, C, D, E, min);

                hipEventRecord(stop);

                hipEventSynchronize(stop);
                milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                printf("Time (ms): %f\n", milliseconds);
                avg += milliseconds;
        }

        printf("min[0]: %f \n",min[0]);
        printf("avgTime (ms): %f\n", avg / testruns);

        std::cout << "######################################################" << std::endl;
        std::cout << "######################################################" << std::endl;

        printf("Speedup Tcpu/Tgpu: %f \n", avg_cpu / avg);
}


int main(void)
{
        int N = (1 << 20);
        std::cout << "N = " << N << std::endl;
        dry_run(N);

}
